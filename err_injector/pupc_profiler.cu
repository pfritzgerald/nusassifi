#include "hip/hip_runtime.h"
/*********************************************************************************** \
* Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
\***********************************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <unistd.h>
#include <assert.h>
#include <string>
#include <map>
#include <fstream>
#include <sys/time.h>
#include <cupti.h>

#include <sassi/sassi-core.hpp>
#include <sassi/sassi-regs.hpp>
#include <sassi/sassi-memory.hpp>
#include "sassi/sassi-opcodes.h"
#include "sassi_intrinsics.h"
#include "sassi_dictionary.hpp"
#include "sassi_lazyallocator.hpp"

#define MAX_FN_STR_LEN 64
//__managed__ sassi::dictionary<int64_t, uint64_t> *kname_map;
//__managed__ int fname_counter;
struct PUPC
{
	uint64_t pupc;
	int basic_block_id;
	char fnName[MAX_FN_STR_LEN];
	SASSIInstrOpcode opcode;
	int32_t numGPRDsts;
	int32_t numGPRSrcs;
	int GPRDsts[SASSI_NUMGPRDSTS];
	int GPRSrcs[SASSI_NUMGPRSRCS];
	bool isMem;
	unsigned long long weight;
};

std::ofstream pupc_ofs;
__managed__ sassi::dictionary<uint64_t, PUPC> *sassi_pupcs;  

///////////////////////////////////////////////////////////////////////////////////
///
///  A simple string copy to copy from device memory to our UVM malloc'd region.
///
///////////////////////////////////////////////////////////////////////////////////
__device__ void simple_strncpy(char *dest, const char *src)
{
  int i;
  for (i = 0; i < MAX_FN_STR_LEN-1; i++) {
    char c = src[i];
    if (c == 0) break;
    dest[i] = c;
  }
  dest[i] = '\0';
}

// output profile filenames for global and non-global memory accesses
std::string pupcs_filename = "pupcs.txt";

// This function will be called after every SASS instruction gets executed 
__device__ void sassi_after_handler(SASSIAfterParams* ap, SASSIRegisterParams *rp) {
	//	printf("FRITZ::::Will Execute\n");
	uint64_t pupc = ap->GetPUPC();
	PUPC *pc_entry = sassi_pupcs->getOrInit(pupc, [ap, pupc, rp] (PUPC* inst) {
		inst->basic_block_id = ap->GetBBID();
		inst->pupc = pupc;
/*		int current_fn_counter = fname_counter;
		uint64_t kname_code = kname_map->getOrInit((int64_t)(ap->GetFnName()), 
						[current_fn_counter] (int knameCode) {
			knameCode = current_fn_counter;
			current_fn_counter += 1;
		};
		fname_counter = current_fn_counter;*/
 
		simple_strncpy(inst->fnName, ap->GetFnName());
		//inst->fnName = kname_code;
		inst->opcode = ap->GetOpcode();
		inst->isMem = ap->IsMem();
		inst->numGPRDsts = rp->GetNumGPRDsts();
		inst->numGPRSrcs = rp->GetNumGPRSrcs();
		for (int i=0; i<rp->GetNumGPRSrcs(); i++)
			inst->GPRSrcs[i] = rp->GetRegNum(rp->GetGPRSrc(i));
		for (int i=0; i<rp->GetNumGPRDsts(); i++)
			inst->GPRDsts[i] = rp->GetRegNum(rp->GetGPRDst(i));
	});
	atomicAdd(&(pc_entry->weight), 1);
}

static void sassi_init()
{
	pupc_ofs.open(pupcs_filename.c_str(), std::ofstream::out);
	sassi_pupcs = new sassi::dictionary<uint64_t, PUPC>();
	//kname_map = new sassi::dictionary<int64_t, uint64_t>();
	//fname_counter = 0;
}

static void sassi_finalize(sassi::lazy_allocator::device_reset_reason reason)
{
	hipDeviceSynchronize();

	sassi_pupcs->map([](uint64_t &k, PUPC &inst) {
		pupc_ofs << "PUPC," << std::hex << inst.pupc << ",BBId,"
		<< std::dec << inst.basic_block_id 
		<< ",fnName," << inst.fnName << ",opcode," 
		<< SASSIInstrOpcodeStrings[inst.opcode] << ",isMem," << inst.isMem << ",weight," 
		<< inst.weight << ",numGPRSrcs," << inst.numGPRSrcs <<",GPRSrcs,";
		for (int i=0; i<inst.numGPRSrcs; i++)
			pupc_ofs << std::dec << inst.GPRSrcs[i] << ",";
		pupc_ofs << "numGPRDsts," << inst.numGPRDsts << ",GPRDsts,";
		for (int i=0; i<inst.numGPRDsts; i++)
			pupc_ofs << std::dec << inst.GPRDsts[i] << ",";
		pupc_ofs << "\n";
	});
/*	kname_map->map([](int64_t &k, uint64_t &code) {
		pupc_ofs << "kernel," << (char *)k << ",code," << code << "\n";
	});*/
	pupc_ofs.close();
}

static sassi::lazy_allocator profilerInit(sassi_init, sassi_finalize); 
