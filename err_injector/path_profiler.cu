#include "hip/hip_runtime.h"
/*********************************************************************************** \
 * Copyright (c) 2015, NVIDIA open source projects
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 * - Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 * 
 * - Neither the name of SASSI nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * This example shows how to use SASSI to inspect the control flow graph.
 *
 * The application code the user instruments should be instrumented with the
 * following SASSI flags: -Xptxas --sassi-function-entry -Xptxas --sassi-bb-entry
 *  
\***********************************************************************************/

#include <algorithm>
#include <assert.h>
#include <map>
#include <fstream>
#include <cupti.h>
#include <stdint.h>
#include <stdio.h>
#include <string>
#include "sassi_intrinsics.h"
#include "sassi_lazyallocator.hpp"
#include "sassi_dictionary.hpp"
#include <sassi/sassi-function.hpp>
#include <sassi/sassi-core.hpp>
#include <sassi/sassi-regs.hpp>


// 8Mb of space for CFG information.
#define POOLSIZE (1024 * 1024 * 1024)
#define MAX_FN_STR_LEN 64
#define MAX_KNAME_SIZE 200
#define MAX_NUM_BBS	500
#define MAX_PATH_NUMBER 200
// flatten thread id
__device__ int get_flat_tid() {
	int tid_b = threadIdx.x + (blockDim.x * (threadIdx.y + 
  				(threadIdx.z * blockDim.y))); // thread id within a block
	int bid = blockIdx.x + (gridDim.x * (blockIdx.y + 
  				(blockIdx.z * gridDim.y))); // block id 
	int tid = tid_b + (bid * blockDim.x * blockDim.y * blockDim.z);
	return tid;
}

typedef struct {
	char kernelName[MAX_KNAME_SIZE];
	uint64_t bb_increments[500];
} path_sum_info_t;

struct PathDesc {
	uint32_t pathId;
	uint16_t BBStart;
	uint16_t BBEnd;
};

//static __managed__ unsigned int interval_size;
//__managed__ unsigned long long AppDynInstCounter;
__managed__ path_sum_info_t path_sums[50];
__managed__ int current_kname_index;
__managed__ uint32_t last_bb_executed[1024*1024]; //[..15b'0..1b'ExitBB?..|..16b'BBID..]
__managed__ int max_path_id;

__managed__ sassi::dictionary<int64_t, PathDesc> *path_tracker;
__managed__ sassi::dictionary<int64_t, unsigned long long> *path_count;

__managed__ sassi::dictionary<int64_t, int64_t*> *full_path_tracker;
__managed__ uint16_t full_path_index[1024*1024];

std::map<std::string, int> knameIndexMap;

void parse_params(std::string filename) {
	std::ifstream ifs (filename.c_str(), std::ifstream::in);
	int num_kernels = 0;
	if (ifs.is_open())
	{
		ifs >> num_kernels;
		printf("num kernels: %d\n", num_kernels);
		assert(num_kernels > 0);
		//path_sums = (path_sum_info_t *)malloc(sizeof(path_sum_info_t) * num_kernels);
		for (int k_idx=0; k_idx<num_kernels; k_idx++)
		{
			int num_bbs = 0;
			ifs >> path_sums[k_idx].kernelName;
			printf("  kName %s\n", path_sums[k_idx].kernelName);
			knameIndexMap[path_sums[k_idx].kernelName] = k_idx;
			ifs >> num_bbs;
			printf("   num bbs: %d\n", num_bbs);
			assert(num_bbs > 0);
			//path_sums[k_idx].bb_increments = (uint64_t *)malloc(sizeof(uint64_t) * num_bbs);
			uint16_t last_bb_from = 65535;
			for (int bb_idx=0; bb_idx<num_bbs; bb_idx++)
			{
				uint16_t bb_from = 65535, bb_to = 65535, inc_val = 65535;
				ifs >> bb_from >> bb_to >> inc_val;
				printf("    from %u to %u, inc %u\n", bb_from, bb_to, inc_val);
			//	bb_increments[x] = [64-48:bb_to_2, 47-32:inc_2, 31-16:bb_to, 15-0:inc]
			//	|63................48|47................32|31................16|15................0|
			//	|--------------------|--------------------|--------------------|-------------------|
			//	|      NEXT BB       |     INC FOR BB     |       NEXT BB      |     INC FOR BB    |
			//	|--------------------|--------------------|--------------------|-------------------|
			//	|---------2ND DIVERGENCE (IF ANY)---------|-----------DEFAULT NEXT PATH------------|
				if(last_bb_from != bb_from){
					path_sums[k_idx].bb_increments[bb_from] = ((uint64_t)(bb_to) << 16) | inc_val;
/*					uint16_t bbnext = (uint16_t) (((1 << 16)-1) & (path_sums[k_idx].bb_increments[bb_from] >> 16));
					uint16_t inc = (uint16_t) (((1 << 16) -1) & (path_sums[k_idx].bb_increments[bb_from]));
					printf(" 1--  from %u to %u, inc %u\n", bb_from, bbnext, inc); */
				}
				else {
					uint64_t tmp = path_sums[k_idx].bb_increments[bb_from];
					path_sums[k_idx].bb_increments[bb_from] = 
							((uint64_t)(bb_to) << 48) | ((uint64_t)inc_val << 32) | tmp;
/*					uint16_t bbnext = (uint16_t) (((1 << 16)-1) & (path_sums[k_idx].bb_increments[bb_from] >> 48));
					uint16_t inc = (uint16_t) (((1 << 16) -1) & (path_sums[k_idx].bb_increments[bb_from] >> 32));
					printf("  2-- from %u to %u, inc %u\n", bb_from, bbnext, inc); */

				}

				last_bb_from = bb_from;
			}
		}
	}
}
		
// Create a memory pool that we can populate on the device and read on the host.
static __managed__ uint8_t sassi_mempool[POOLSIZE];
static __managed__ int     sassi_mempool_cur;

std::ofstream bb_ofs;
std::ofstream fp_ofs;

// A dictionary of paths per thread.
//static __managed__ sassi::dictionary<int64_t, PATH_TRACKER> *sassi_path;

// A dictionary of counts for each path.
//static __managed__ sassi::dictionary<int64_t, PATH_COUNT> *sassi_path_count;

///////////////////////////////////////////////////////////////////////////////////
///
///  Allocate data from the UVM mempool.
///
///////////////////////////////////////////////////////////////////////////////////
__device__ void *simple_malloc(size_t sz)
{
  int ptr = atomicAdd(&sassi_mempool_cur, sz);
  assert ((ptr + sz) <= POOLSIZE);
  return (void*) &(sassi_mempool[ptr]);
}

///////////////////////////////////////////////////////////////////////////////////
///
///  A simple string copy to copy from device memory to our UVM malloc'd region.
///
///////////////////////////////////////////////////////////////////////////////////
__device__ void simple_strncpy(char *dest, const char *src)
{
  int i;
  for (i = 0; i < MAX_FN_STR_LEN-1; i++) {
    char c = src[i];
    if (c == 0) break;
    dest[i] = c;
  }
  dest[i] = '\0';
}
__device__ int simple_strncmp(char *dest, const char *src)
{
  int i;
  for (i = 0; i < MAX_FN_STR_LEN-1; i++) {
    if (dest[i] != src[i])
	    return -1;
  //  if ((dest[i] == 0) && (src[i] == 0))
//    else if ((dest[i] == 0) || (src[i] == 0))
//	    return -2;
  }
  
	    return 0;
}
///////////////////////////////////////////////////////////////////////////////////
///
///  A call to this function will be inserted at the beginning of every 
///  CUDA function or kernel.  We will essentially perform a deep copy of the
///  CFG SASSI presents.  All of the copied data only contains static information
///  about the CFG.  In the sassi_basic_block_entry handler, below, we will 
///  record the dynamic number of times the basic block is invoked.
///
///////////////////////////////////////////////////////////////////////////////////
/*
__device__ void sassi_function_entry(SASSIFunctionParams* fp)
{

}
*/

///////////////////////////////////////////////////////////////////////////////////
///
///  Simply lookup the basic block in our dictionary, get its "weight" feild
///  and increment it.
///
///////////////////////////////////////////////////////////////////////////////////
__device__ void sassi_basic_block_entry(SASSIBasicBlockParams *bb)
{
	if (simple_strncmp(path_sums[current_kname_index].kernelName,bb->GetFnName()) !=0)
		return;
	int threadId = get_flat_tid();
	bool is_backedge = false;
	if ((threadId % 32) == 0) {
		int64_t path_tracker_idx = threadId/32;
		int bb_id = bb->GetID();
		uint16_t last_bb_id = (uint16_t)(last_bb_executed[path_tracker_idx] & 0xffff);

		PathDesc *tracker = path_tracker->getOrInit(path_tracker_idx, [bb_id](PathDesc *pd) {
		pd->pathId = 0;
		pd->BBStart = (uint16_t)bb_id;
		pd->BBEnd = 65535;
		});

	//printf("threadID: %d reporting\n", threadId);
	//FIXME:: What if somehow this entry already existed?
		// Checking for self loop
		if ((last_bb_id == (uint16_t)bb_id) && (!bb->IsEntryBlock())) // because last_bb is set to 0 at the kernel exit
		{
			// we have a self loop
			int64_t path_key = ((int64_t)(0) << 32) | ((uint16_t)(bb_id) << 16) | (uint16_t)bb_id;
			unsigned long long* PathCount = path_count->getOrInit(path_key, [] (unsigned long long *count) {
				*count = 0;
			});
			atomicAdd(PathCount, 1);

			int64_t* (full_path) = *(full_path_tracker->getOrInit(path_tracker_idx, [path_key](int64_t **(full_path)) {
				int64_t *path = (int64_t*)simple_malloc(150*sizeof(int64_t));
				path[0] = path_key;
				*full_path = path;
			}));
			full_path[full_path_index[path_tracker_idx]] = path_key;
			full_path_index[path_tracker_idx] += 1;

		}
		// We have not exited the kernel, check if last BB executed was an exit BB
		if ((last_bb_executed[path_tracker_idx] >> 16) == 0x1)
		{
			uint16_t path_id = (tracker)->pathId;
			__threadfence();
			int64_t path_key = ((int64_t)(path_id) << 32) | ((tracker)->BBStart<<16) | (uint16_t)last_bb_id;
			unsigned long long* PathCount = path_count->getOrInit(path_key, [] (unsigned long long *count) {
				assert(0); // it should have existed already
			});
			atomicAdd(PathCount, -1);

			int64_t* (full_path) = *(full_path_tracker->getOrInit(path_tracker_idx, [path_key](int64_t **(full_path)) {
			}));
			full_path[full_path_index[path_tracker_idx]] = 65535;
			full_path_index[path_tracker_idx] -= 1;

		}

		// update path value based on inc_val
		uint64_t bbNext_and_Inc = path_sums[current_kname_index].bb_increments[last_bb_id];
		uint16_t bbNext = (uint16_t) (((1 << 16)-1) & (bbNext_and_Inc >> 16));
		uint16_t inc = 0;
		if (bb_id == bbNext)
			inc = (uint16_t) (((1 << 16) -1) & (bbNext_and_Inc));
		else
		{
			bbNext = (uint16_t) (((1 << 16) -1) & (bbNext_and_Inc >> 48));
			if (bb_id == bbNext) {
				inc = (uint16_t) (((1 << 16) -1) & (bbNext_and_Inc >> 32));
			}
			else if (!bb->IsEntryBlock())// can't find an edge from last bb to this bb, it must be a backedge
			{
				is_backedge = true;
			}
		}
		if (!is_backedge)
			(tracker)->pathId += inc;
		else
		{
//			printf("--------------BACK EDGE: BB%u->BB%u ------------------\n", last_bb_id, bb_id);
			uint32_t path_id = (tracker)->pathId;
			__threadfence();
			if (path_id > max_path_id)
				max_path_id = path_id;
			// PATH KEY:
			//|..........32 bits..........|...16 bits...|...16 bits...|
			//|---------------------------|-------------|-------------|
			//|          PATH ID          |   BBStart   |    BBEnd    |
			//|---------------------------|-------------|-------------|
			int64_t path_key = ((int64_t)(path_id) << 32) | ((tracker)->BBStart<<16) | (uint16_t)last_bb_id;
			unsigned long long* PathCount = path_count->getOrInit(path_key, [] (unsigned long long *count) {
				*count = 0;
			});
			atomicAdd(PathCount, 1LL);
			(tracker)->pathId = 0;
			(tracker)->BBStart = (uint16_t)bb_id;
			(tracker)->BBEnd = 65535;

			int64_t* (full_path) = *(full_path_tracker->getOrInit(path_tracker_idx, [path_key](int64_t **(full_path)) {
				int64_t *path = (int64_t*)simple_malloc(150*sizeof(int64_t));
				path[0] = path_key;
				*full_path = path;
			}));
			full_path[full_path_index[path_tracker_idx]] = path_key;
			full_path_index[path_tracker_idx] += 1;

		}
	
		last_bb_executed[path_tracker_idx] = 0|(uint16_t)bb_id;
		if (bb->IsExitBlock())
		{
			// A kernel may have multiple exits, therefore the kernel may not exit at this basic block.
			// This is why we keep track of the BB that was marked as an exit
			last_bb_executed[path_tracker_idx] = (0x1<<16) | (uint16_t)(bb_id);
			//printf("exit block\n");
			uint16_t path_id = (tracker)->pathId;
			__threadfence();
			if (path_id > max_path_id)
				max_path_id = path_id;
			int64_t path_key = ((int64_t)(path_id) << 32) | ((tracker)->BBStart<<16) | (uint16_t)bb_id;
			//printf("path_id: %u, BBStart %u, BBEnd %u\n",
			//	path_id, (tracker)->BBStart, bb_id);
			unsigned long long* PathCount = path_count->getOrInit(path_key, [] (unsigned long long *count) {
					*count = 0;
				});
				atomicAdd(PathCount, 1LL);

			int64_t* (full_path) = *(full_path_tracker->getOrInit(path_tracker_idx, [path_key](int64_t **(full_path)) {
				int64_t *path = (int64_t*)simple_malloc(150*sizeof(int64_t));
				path[0] = path_key;
				*full_path = path;
			}));
			full_path[full_path_index[path_tracker_idx]] = path_key;
			full_path_index[path_tracker_idx] += 1;

		}
	}
}


// This function will be exected before a kernel is launced
static void onKernelEntry(const CUpti_CallbackData *cbInfo) {
	hipError_t * error = (hipError_t*) cbInfo->functionReturnValue;

	if ( (*error) != hipSuccess ) {
		printf("Kernel Entry Error: %d\n", (*error));
	}
	path_tracker->clear();
	hipDeviceSynchronize();
	std::string kName = cbInfo->symbolName; // name of kernel
	bb_ofs << "kernel," << kName << "\n";
	fp_ofs << "kernel," << kName << "\n";
	current_kname_index = knameIndexMap[kName];
	printf("kernel %s: index: %d\n", kName.c_str(), current_kname_index);

}

// This function will be exected after the kernel exits 
static void onKernelExit(const CUpti_CallbackData *cbInfo) {
	hipDeviceSynchronize();
	hipError_t *error = (hipError_t*) cbInfo->functionReturnValue;
	if ((*error) != hipSuccess) {
		printf("Kernel Exit error: %d\n", (*error));
	}
	std::string kName = cbInfo->symbolName;
	//printf("printing path profile after %s\n", kName.c_str());
	path_count->map([kName](int64_t k, unsigned long long &c) {
		//printf("PATH  => %lld\n", c);//(uint16_t)(k>>48), *c);
		bb_ofs << "kname," << kName << ",path_id," << (uint32_t) (k>>32) << ",BBStart,"
			<< (uint16_t)((k>>16)&(0x00ff)) << ",BBEnd," <<(uint16_t)((k)&(0x000000ff)) <<",count,"
			<< c << "\n";
	});
	full_path_tracker->map([kName](int64_t k, int64_t* &c) {
		//printf("PATH  => %lld\n", c);//(uint16_t)(k>>48), *c);
		fp_ofs << "WARP " << k << "=>";
		for (int i=0; i<full_path_index[k]; i++)
			fp_ofs<< (uint32_t) (c[i]>>32) << ":"<< (uint16_t)((c[i]>>16)&(0x00ff)) << "-"
			<<(uint16_t)((c[i])&(0x000000ff)) << ">";
		fp_ofs << "\n";
//			 <<",count,"
//			<< c << "\n";
	});

	bzero(last_bb_executed, sizeof(last_bb_executed));
	bzero(full_path_index, sizeof(full_path_index));
	path_count->clear();
	full_path_tracker->clear();
	sassi_mempool_cur = 0;
	bzero(sassi_mempool, sizeof(sassi_mempool));
}
///////////////////////////////////////////////////////////////////////////////////
///
///  Print the graph out in "dot" format.  
///  E.g., use:
///
///       dot -Tps -o graph.ps sassi-cfg.dot 
///
///  to render the graph in postscript.
///
///////////////////////////////////////////////////////////////////////////////////
static void sassi_finalize(sassi::lazy_allocator::device_reset_reason unused)
{
	hipDeviceSynchronize();
	bb_ofs.close();
	fp_ofs.close();
}

///////////////////////////////////////////////////////////////////////////////////
///
///  Initialize the UVM memory pool and our two dictionaries.  
///
///////////////////////////////////////////////////////////////////////////////////
static void sassi_init()
{
	parse_params("cfgs.txt");
	bb_ofs.open("path_profile.txt", std::ofstream::out);
	fp_ofs.open("full_paths.txt", std::ofstream::out);
//	bzero(path_count, sizeof(path_count));
	path_tracker = new sassi::dictionary<int64_t, PathDesc>();
	path_count = new sassi::dictionary<int64_t, unsigned long long>();
	full_path_tracker = new sassi::dictionary<int64_t, int64_t*>();
//	AppDynInstCounter = 0;
	current_kname_index = 0;
	max_path_id = 0;
	bzero(last_bb_executed, sizeof(last_bb_executed));
	bzero(full_path_index, sizeof(full_path_index));
	sassi_mempool_cur = 0;
	bzero(sassi_mempool, sizeof(sassi_mempool));
}


///////////////////////////////////////////////////////////////////////////////////
///
///  
///
///////////////////////////////////////////////////////////////////////////////////
static sassi::lazy_allocator mapAllocator(sassi_init, sassi_finalize, onKernelEntry, onKernelExit);
